#include "hip/hip_runtime.h"
//#include "kernel.cuh"
#include <stdio.h>
#include <stdlib.h> 

#include "hip/hip_runtime.h"
#include ""

int GenerateRand(int rmax)
{
	return ((int)rand() / (int)(RAND_MAX)) * rmax;
}

void GenerateVector(int *v, int vecSz)
{
	for (int i = 0; i < vecSz; i++)
		v[i] = GenerateRand(100);
}

__global__
void AddInt_Kernel(int *a_d, int *b_d, int *c_d, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n)
		c_d[i] = a_d[i] + b_d[i];
}

hipError_t CudaSetup(const int *a, const int *b, int *c, int vecSz)
{
	int *a_d = 0;
	int *b_d = 0;
	int *c_d = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		printf("ERROR: hipSetDevice failed!\n");

	cudaStatus = hipMalloc((void**)&c_d, vecSz * sizeof(int));
	cudaStatus = hipMalloc((void**)&a_d, vecSz * sizeof(int));
	cudaStatus = hipMalloc((void**)&b_d, vecSz * sizeof(int));

	cudaStatus = hipMemcpy(a_d, a, vecSz * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(b_d, b, vecSz * sizeof(int), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess)
		printf("ERROR");

	//Kernel launch
	dim3 DimGrid(vecSz / 256, 1, 1);
	if (vecSz % 256) DimGrid.x++;
	dim3 DimBlock(256, 1, 1);
	AddInt_Kernel<<<DimGrid, DimBlock>>>(a_d, b_d, c_d, vecSz); //Unsure about DimGrid and Dimblock
	//AddInt_Kernel << <ceil(vecSz / 256), 256 >> >(a_d, b_d, c_d, vecSz);

	//AddInt_Kernel<<<1, vecSz >>>(a_d, b_d, c_d, vecSz);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(c, c_d, vecSz * sizeof(int), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	return cudaStatus;
}

int main()
{
	srand((unsigned int)time(NULL));

	const int vecSz = 5;
	/*int *a = (int*)malloc(vecSz * sizeof(int));
	int *b = (int*)malloc(vecSz * sizeof(int));
	int *c = NULL;*/

	const int a[vecSz] = { 1, 2, 3, 4, 5 };
	const int b[vecSz] = { 10, 20, 30, 40, 50 };
	int c[vecSz] = { 0 };

	/*GenerateVector(a, vecSz);
	GenerateVector(b, vecSz);*/

	CudaSetup(a, b, c, vecSz);

	printf("C result:");
	for (int i = 0; i < vecSz; i++)
		printf(" %d", c[i]);

	printf("\nDone..\n");

	hipDeviceReset();
	getchar();

	return 0;
}